#include <cstdio>
#include <hip/hip_runtime.h>

#define SIZE 5

#if defined(NDEBUG) // release mode
#define CUDA_CHECK(x)(x)
#else              //debug mode
#define CUDA_CHECK(X) do{\
    (X);\
    hipError_t e = hipGetLastError();\
    if(hipSuccess != e){\
        printf("cuda failure %s at %s : %d", hipGetErrorString(e), __FILE__, __LINE__);\
        exit(1);\
    }\
}while(0)
#endif

int main()
{
    int* dev_a;
    int* dev_b;
	const int a[SIZE] = { 1,2,3,4,5 };
	int b[SIZE] = { 1,2,3,4,5 };
    
    //allocate device memory
    CUDA_CHECK(hipMalloc((void**)&dev_a, SIZE * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&dev_b, SIZE * sizeof(int)));

    //copy from host to device
    //gonna cause error
	CUDA_CHECK(hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyDeviceToDevice));

    //copy from device to host
	CUDA_CHECK(hipMemcpy(b, dev_b, SIZE * sizeof(int), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(dev_a));
    CUDA_CHECK(hipFree(dev_b));
    return 0;
}	